#include "hip/hip_runtime.h"
// һ���򵥵�����Ӧ��ʾ��

#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <>

#define N  10000


__global__ void kernel_1()
{
	double sum = 0;
	for (int i = 0; i < N; i++)
	{
		sum = sum + tan(0.1) * tan(0.1);
	}
}

__global__ void kernel_2()
{
	double sum = 0;
	for (int i = 0; i < N; i++)
	{
		sum = sum + tan(0.1) * tan(0.1);
	}
}

__global__ void kernel_3()
{
	double sum = 0;
	for (int i = 0; i < N; i++)
	{
		sum = sum + tan(0.1) * tan(0.1);
	}
}

__global__ void kernel_4()
{
	double sum = 0;
	for (int i = 0; i < N; i++)
	{
		sum = sum + tan(0.1) * tan(0.1);
	}
}


void kernel_stream()
{
	int dev = 0;
	hipSetDevice(dev);
	const int n_stream = 8;

	// ������
	hipStream_t stream[n_stream];
	for(int i=0; i<n_stream; i++)
	{
		hipStreamCreate(&stream[i]);
	}
	dim3 block(1);
	dim3 grid(1);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	for (int i = 0; i < n_stream; i++)
	{
		kernel_1 << <grid, block, 0, stream[i] >> > ();
		kernel_2 << <grid, block, 0, stream[i] >> > ();
		kernel_3 << <grid, block, 0, stream[i] >> > ();
		kernel_4 << <grid, block, 0, stream[i] >> > ();
	}
	hipEventRecord(stop, 0);

	// ͬ��
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time to do multiple streams: %3.1f ms\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	for (int i = 0; i < n_stream; i++)
	{
		hipStreamDestroy(stream[i]);
	}
	hipDeviceReset();

}