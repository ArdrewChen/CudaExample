#include "hip/hip_runtime.h"
// �߳���ϴ��ָ��
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

# define N 32

__device__ int warpShfl(int value, int lane)
{
	return __shfl_sync(0xFFFFFFFF, value, lane);
}

__device__ int warpShflXor(int value, int lane, int width)
{
	return __shfl_xor_sync(0xFFFFFFFF, value, lane, width);
}

__device__ int warpShflUp(int value, int lane, int width)
{
	return __shfl_up_sync(0xFFFFFFFF, value, lane, width);
}


__device__ int warpShflDown(int value, int lane, int width)
{
	return __shfl_down_sync(0xFFFFFFFF, value, lane, width);
}



__global__ void kernel_wrapshfl(int* in, int* out)
{
	int value = in[threadIdx.x];
	//value = warpShfl(value, 1);
	//value = warpShflXor(value, 1, 32);
	//value = warpShflUp(value, 1, 32);
	value = warpShflDown(value, 1, 32);
	out[threadIdx.x] = value;
}



void kernel_wrapshfl()
{
	// ѡ���豸
	int dev = 0;
	hipSetDevice(dev);
	
	int nBytes = N * sizeof(int);
	int* h_in = (int*)malloc(nBytes);
	int* h_out = (int*)malloc(nBytes);

	int* d_in = NULL;
	int* d_out = NULL;

	hipMalloc((int**)&d_in, nBytes);
	hipMalloc((int**)&d_out, nBytes);

	printf("input data: ");

	for(int i=0; i<N; i++)
	{
		h_in[i] = i;
		printf("%d ", h_in[i]);
	}
	printf("\n");

	dim3 block(N, 1);
	dim3 grid(1, 1);
	hipMemcpy(d_in, h_in, nBytes, hipMemcpyHostToDevice);
	kernel_wrapshfl << <grid, block >> > (d_in,d_out);
	hipMemcpy(h_out, d_out, nBytes, hipMemcpyDeviceToHost);
	hipDeviceReset();

	printf("output data: ");
	for (int i = 0; i < N; i++)
	{
		printf("%d ", h_out[i]);
	}
	printf("\n");
	free(h_in);
	free(h_out);
	hipFree(d_in);
	hipFree(d_out);
}