#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <iostream>
#include "tools.cuh"


__global__ void printThreadIndex()
{	
	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iy = threadIdx.y + blockIdx.y * blockDim.y;
	printf("ix: %d, iy: %d\n", ix, iy);
}

__global__ void sumMatrix_GPU()
{

}

void sumMatrix_CPU()
{

}
void kernel_sumMatrix()
{	
	int dev = 0;
	hipSetDevice(dev);

	//��������С
	int nx = 16;
	int ny = 16;
	int nBytes = nx * ny * sizeof(float);
	
	//�����ڴ�����
	float* h_a = (float*)malloc(nBytes);
	float* h_b = (float*)malloc(nBytes);
	float* h_res = (float*)malloc(nBytes);

	//�豸�ڴ�����
	float* d_a = NULL;
	float* d_b = NULL;
	float* d_res = NULL;
	hipMalloc((float**)&d_a, nBytes);
	hipMalloc((float**)&d_b, nBytes);
	hipMalloc((float**)&d_res, nBytes);

	//��ʼ����������
	initialData(h_a, nx * ny);
	initialData(h_b, nx * ny);	
	
	// ���������ݿ������豸
	hipMemcpy(d_a, h_a, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, nBytes, hipMemcpyHostToDevice);

	//�����߳̿������
	dim3 block(4, 3);
	dim3 grid(2, 1);
	
	// ��ӡ�߳������������߳�������ʽ
	// printThreadIndex<<<grid, block>>>();
	hipDeviceSynchronize();
}