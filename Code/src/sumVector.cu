#include "hip/hip_runtime.h"
// ���һ������������Ԫ�صĺͣ�ʹ�ù�Լ��
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <iostream>
#include "tools.cuh"

// CPUʵ����ͣ���֤�����ȷ��
float sumVector_CPU(float *idata, int n)
{
	float sum = 0;
	for (int i = 0; i < n; i++)
	{
		sum = sum + idata[i];
	}
	return sum;
}

// GPU���й�Լʵ�֣��������֮�ͣ�δ���Ʒֻ����⣬�߳�����Ч�ʵ�
__global__ void sumVector_GPU(float *idata, float *odata, unsigned int size) 
{
	int tid = threadIdx.x;
	float *idata_tmp = idata + blockIdx.x * blockDim.x;

	// forѭ��һ���Ǽ���һ���߳̿��Ԫ��
	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		if (tid % (2 * stride) == 0)              // �ڴ˴�������֧
		{
			idata_tmp[tid] += idata_tmp[tid + stride];
		}
		__syncthreads(); // ͬ�������߳�
	}
	if (tid == 0)
	{
		odata[blockIdx.x] = idata_tmp[0];
	}
}

// ���Ʒֻ�1���߳�������������������һһ��Ӧ
__global__ void sumVectorImproved1_GPU(float *idata, float *odata, unsigned int size) 
{
	int tid = threadIdx.x;
	float *idata_tmp = idata + blockIdx.x * blockDim.x;
	for(int stride=1; stride<blockDim.x; stride *= 2)
	{
		int index = 2 * stride * tid;	// �ؼ����裬�����������������ӳ�䵽�߳����������Ǹ����߳����������ƶ������ٷֻ�
										// ȷ��ǰ�����߳�������	 
		if(index < blockDim.x)
		{
			idata_tmp[index] += idata_tmp[index + stride];
		}
		__syncthreads();
	}
	if(tid == 0)
	{
		odata[blockIdx.x] = idata_tmp[0];
	}

}

// ���Ʒֻ�2����������������ֳɶ���飬ʹ�ö���߳̿鴦�����ݣ��Ƚ���һ�������ӷ���Ȼ���Լ
__global__ void sumVectorImproved2_GPU(float* idata, float* odata, unsigned int size)
{
	int tid = threadIdx.x;
	float* idata_tmp = idata + blockIdx.x * blockDim.x * 2;
	int idx = tid + blockIdx.x * blockDim.x * 2;
	if (idx + blockDim.x < size)		// �ؼ����裬�Ƚ���һ�������ӷ�
	{
		idata[idx] += idata[idx + blockDim.x];
	}
	for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
	{
		if (tid < blockDim.x)
		{
			idata_tmp[tid] += idata_tmp[tid + stride];
		}
		__syncthreads();
	}
	if (tid == 0)
	{
		odata[blockIdx.x] = idata_tmp[0];
	}
}


// Ԥ�Ⱥ���
__global__ void warmup() {
	printf("just warmup!\n");
}

void kernel_sumVector() 
{
	// ѡ���豸
	int dev = 0;
	hipSetDevice(dev);

	//��������С
	int nx = 1024;
	int nBytes = nx  * sizeof(float);

	//�����ڴ�����
	float* h_a = (float*)malloc(nBytes);
	float* h_res_fromGPU = (float*)malloc(nBytes);

	//�豸�ڴ�����
	float* d_a = NULL;
	float* d_res = NULL;
	hipMalloc((float**)&d_a, nBytes);
	hipMalloc((float**)&d_res, nBytes);

	//��ʼ���������ݣ���ʼ������
	initialData(h_a, nx);	
	
	//�����߳̿������
	dim3 block(nx, 1);    // ����ά�ȣ��޷���������
	dim3 grid((nx-1)/ block.x +1, 1);

	// Ԥ��
	warmup<< <1,1 >> > ();
	hipDeviceSynchronize();

	// ���ú˺���
	// ���������ݿ������豸
	hipMemcpy(d_a, h_a, nBytes, hipMemcpyHostToDevice); // δ���Ʒֻ�����
	hipEvent_t start, stop;
	float duration_gpu = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	sumVector_GPU << <grid, block >> > (d_a, d_res, nx);
	hipDeviceSynchronize();

	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&duration_gpu, start, stop);
	std::cout << "sumVector_GPU����ʱ�� = " << duration_gpu << "ms" << std::endl;

	// ���������������
	hipMemcpy(h_res_fromGPU, d_res, nBytes, hipMemcpyDeviceToHost);
	float sum = 0;
	sum = h_res_fromGPU[0];
	std::cout << "sumVector_GPU sum = " << sum << std::endl;


	// ���и��Ʒֻ�1
	// ���������ݿ������豸
	hipMemcpy(d_a, h_a, nBytes, hipMemcpyHostToDevice); // δ���Ʒֻ�����
	hipEvent_t start1, stop1;
	float duration_gpu1 = 0.0000f;
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	hipEventRecord(start1, 0);

	sumVectorImproved1_GPU << <grid, block >> > (d_a, d_res, nx);
	hipDeviceSynchronize();

	hipEventRecord(stop1, 0);
	hipEventSynchronize(start1);
	hipEventSynchronize(stop1);
	hipEventElapsedTime(&duration_gpu1, start1, stop1);
	std::cout << "sumVectorImproved1_GPU����ʱ�� = " << duration_gpu1 << "ms" << std::endl;

	// ���������������
	hipMemcpy(h_res_fromGPU, d_res, nBytes, hipMemcpyDeviceToHost);
	float sum1 = 0;
	sum1 = h_res_fromGPU[0];
	std::cout << "sumVectorImproved_GPU sum = " << sum1 << std::endl;


	// ���и��Ʒֻ�2������2��Ҫ����飬��Ҫ����block��grid��ֵ
	int block_n = 2; // ���ֵĿ���
	dim3 block2(nx/block_n, 1);
	dim3 grid2((nx/block_n - 1) / block.x + 1, 1);
	hipMemcpy(d_a, h_a, nBytes, hipMemcpyHostToDevice); 
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	hipEventRecord(start1, 0);

	sumVectorImproved2_GPU << <grid2, block2 >> > (d_a, d_res, nx);
	hipDeviceSynchronize();

	hipEventRecord(stop1, 0);
	hipEventSynchronize(start1);
	hipEventSynchronize(stop1);
	hipEventElapsedTime(&duration_gpu1, start1, stop1);
	std::cout << "sumVectorImproved2_GPU����ʱ�� = " << duration_gpu1 << "ms" << std::endl;

	// ���������������
	hipMemcpy(h_res_fromGPU, d_res, nBytes, hipMemcpyDeviceToHost);
	sum1 = h_res_fromGPU[0];
	std::cout << "sumVectorImproved2_GPU sum = " << sum1 << std::endl;


	// CPUִ��
	float sum_cpu = 0;
	sum_cpu = sumVector_CPU(h_a, nx);
	std::cout << "CPU sum = " << sum_cpu << std::endl;



	// �ͷ��ڴ�
	hipFree(d_a);
	hipFree(d_res);


	free(h_a);
	free(h_res_fromGPU);

}