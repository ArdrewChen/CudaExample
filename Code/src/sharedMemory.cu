#include "hip/hip_runtime.h"
// �����ڴ�Ĵ洢�����ξ���
// �Ż�Ŀ�꣬���⹲���ڴ�洢���ͻ
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>

#include "tools.cuh"

#define N  32


// ��ȫ���ڴ�����ݿ����������ڴ�,������
__global__ void sharedMemoryRow(float* out,float* in)
{
	__shared__ float shared[N][N];
	unsigned int ix = threadIdx.x + threadIdx.y * blockDim.x;
	shared[threadIdx.y][threadIdx.x] = in[ix];
	__syncthreads();
	out[ix] = shared[threadIdx.y][threadIdx.x];
}



// ��ȫ���ڴ�����ݿ����������ڴ�,������
__global__ void sharedMemoryCol(float* out,float* in)
{
	__shared__ float shared[N][N];
	unsigned int ix = threadIdx.x + threadIdx.y * blockDim.x;
	shared[threadIdx.x][threadIdx.y] = in[ix];
	__syncthreads();
	out[ix] = shared[threadIdx.x][threadIdx.y];
}

// ��ȫ���ڴ�����ݿ�������̬�����ڴ�,������
__global__ void sharedMemoryRowDynamic(float* out, float* in)
{
	extern __shared__ float shared[];  //extern�ؼ��֣���ʾ��̬�����ڴ�,
	unsigned int ix = threadIdx.x + threadIdx.y * blockDim.x;
	shared[threadIdx.y * N + threadIdx.x] = in[ix];
	__syncthreads();
	out[ix] = shared[threadIdx.y * N + threadIdx.x];
}


void kernel_sharedMemory() 
{
	// ѡ���豸
	int dev = 0;
	hipSetDevice(dev);


	int nBytes = N * N * sizeof(float);
	float* h_in = (float*)malloc(nBytes);	
	float* h_out_row = (float*)malloc(nBytes);
	float* h_out_col = (float*)malloc(nBytes);

	float* d_in = NULL;
	float* d_out_row = NULL;
	float* d_out_col = NULL;
	hipMalloc((float**)&d_in, nBytes);
	hipMalloc((float**)&d_out_row,nBytes);
	hipMalloc((float**)&d_out_col, nBytes);


	dim3 block(N, N);
	dim3 grid(1, 1);
	initialData(h_in, N * N);
	hipMemcpy(d_in, h_in, nBytes, hipMemcpyHostToDevice);

	sharedMemoryRow << <grid, block >> >(d_out_row,d_in);
	sharedMemoryCol << <grid, block >> >(d_out_col,d_in);
	sharedMemoryRowDynamic << <grid, block, nBytes >> >(d_out_row, d_in); // �˺�����Ҫָ����̬�����ڴ��С

	hipMemcpy(h_out_row, d_out_row, nBytes, hipMemcpyDeviceToHost);
	hipMemcpy(h_out_col, d_out_row, nBytes, hipMemcpyDeviceToHost);
	hipDeviceReset();

	// ��ӡ���
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%.2f ", h_out_row[i * N + j]);
		}
		printf("\n");
	}
	free(h_in);
    free(h_out_row);
	free(h_out_col);
	hipFree(d_in);
    hipFree(d_out_row);
	hipFree(d_out_col);

}