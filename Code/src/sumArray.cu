#include "hip/hip_runtime.h"
/*ʵ�������������*/
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <iostream>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

// GPU�˺�����ʵ���������
__global__ void sumArrayGPU(float* d_a, float* d_b, float* d_res)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	d_res[i] = d_a[i] + d_b[i];
}

// CPU������ʵ���������
void sumArrays(float* a, float* b, float* res, const int size)
{
	for (int i = 0; i < size; i++)
	{
		res[i] = a[i] + b[i];
	}
}

// ��֤����Ƿ���ȷ
void checkResult(float* res, float* res_from_gpu, const int size)
{
	for (int i = 0; i < size; i++)
	{
		if (res[i] != res_from_gpu[i])
		{
			printf("Error: %d element do not match!\n", i);
		}
	}
	printf("Check result success!\n");
}

// �����������
void initialData(float* ip, int size)
{
	// generate different seed for random number
	time_t t;
	srand((unsigned)time(&t));
	for (int i = 0; i < size; i++)
	{
		ip[i] = (float)(rand() & 0xFF) / 10.0f;
	}
	return;
}

void kernel_sumArray()
{
	int dev = 0;
	hipSetDevice(dev);

	int nElem = 32;
	int nByte = nElem * sizeof(float);
	float* h_a = (float*)malloc(nByte);
	float* h_b = new float[nElem];
	float* h_res = (float*)malloc(nByte);
	float* h_res_from_gpu = new float[nElem];
	memset(h_res, 0, nByte);
	memset(h_res_from_gpu, 0, nByte);

	float* d_a, * d_b, * d_res;
	hipMalloc((float**)&d_a, nByte);						//�����豸�ڴ�
	hipMalloc((float**)&d_b, nByte);
	hipMalloc((float**)&d_res, nByte);

	initialData(h_a, nElem);
	initialData(h_b, nElem);

	hipMemcpy(d_a, h_a, nByte, hipMemcpyHostToDevice);   //��h_a������d_a
	hipMemcpy(d_b, h_b, nByte, hipMemcpyHostToDevice);

	dim3 block(nElem / 4);
	dim3 grid(nElem / block.x);

	sumArrayGPU << <grid, block >> > (d_a, d_b, d_res);  // ��ʱǰ��ΪԤ�Ⱥ���

	// ��Ӽ�ʱ��
	hipEvent_t start, stop;
	float duration_gpu = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	sumArrayGPU << <grid, block >> > (d_a, d_b, d_res);  // ִ�к˺���

	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&duration_gpu, start, stop);
	std::cout << "����ʱ�� = " << duration_gpu << "ms" << std::endl;

	printf("Execution configuration<<<%d,%d>>>\n", block.x, grid.x);
	hipMemcpy(h_res_from_gpu, d_res, nByte, hipMemcpyDeviceToHost);
	sumArrays(h_a, h_b, h_res, nElem);

	checkResult(h_res, h_res_from_gpu, nElem);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_res);			//�ͷ��豸�ڴ�

	free(h_a);
	free(h_b);
	free(h_res);
	free(h_res_from_gpu);		//�ͷ������ڴ�

	hipDeviceReset();
}